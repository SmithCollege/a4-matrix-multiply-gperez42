#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "hipblas.h" 

// CUDA runtime


#define SIZE 128

// Resources Used: https://www.javatpoint.com/how-to-add-matrix-in-c

int main() {
	 printf("[Matrix Multiply CUBLAS] - Starting...\n");

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	
	const float alpha = 1.0f;
	const float beta = 0.0f;
       
	int size = 100;

	float *x, *y, *z;
	hipMallocManaged(&x, SIZE*sizeof(float) * size * size);
	hipMallocManaged(&y, SIZE*sizeof(float) * size * size);
	hipMallocManaged(&z, SIZE*sizeof(float) * size * size);

  	for (int i = 0; i < size; i++) {
    	for (int j = 0; j < size; j++) {
	      	x[i * size + j] = 1; // x[i][j]
	      	y[i * size + j] = 1;
    	}
  	}

  // MatrixMulOnDevice<<<1,SIZE>>>(x,y,z,size);
  hipblasSgemm(
	  handle, HIPBLAS_OP_N, HIPBLAS_OP_N, size, size,
      size, &alpha, x, size, y, size, &beta, z, size);

  printf("%s\n", hipGetErrorString(hipGetLastError()));

  // synchronize 
  hipDeviceSynchronize();

  for (int i = 0; i < size; i++) {
  	for (int j = 0; j < size; j++) {
  		printf("%f ", z[i * size + j]);
  		if (z[i * size + j] != size) {
  			printf("Error at z[%d][%d]: %f\n", i, j, z[i * size + j]);
  		}
    }
    printf("\n");
  }

  	// Freeing memory
	hipFree(x);
	hipFree(y);
	hipFree(z);
  

  return 0;
}