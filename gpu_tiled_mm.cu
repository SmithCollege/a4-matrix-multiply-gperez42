#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define SIZE 128
#define TILE_WIDTH 2

// Resources Used: https://www.javatpoint.com/how-to-add-matrix-in-c

__global__ void MatrixMulOnDevice(float* A, float* B, float* C, int Width) {
	 for (int i = 0; i < Width; ++i){
		 for (int j = 0; j < Width; ++j) {
			 float sum = 0;
			 for (int k = 0; k < Width; ++k) {
				 float a = A[i * Width + k];
				 float b = B[k * Width + j];
				 sum += a * b;
			 }
		 C[i * Width + j] = sum;

		 }
	 }
}

int main() {
	int size = 100;

	float *x, *y, *z;
	hipMallocManaged(&x, SIZE*sizeof(float) * size * size);
	hipMallocManaged(&y, SIZE*sizeof(float) * size * size);
	hipMallocManaged(&z, SIZE*sizeof(float) * size * size);

  	for (int i = 0; i < size; i++) {
    	for (int j = 0; j < size; j++) {
	      	x[i * size + j] = 1; // x[i][j]
	      	y[i * size + j] = 1;
    	}
  	}
  

  // MatrixMulOnDevice<<<1,SIZE>>>(x,y,z,size);
  MatrixMulOnDevice<<<pow((size/TILE_WIDTH), 2), pow(TILE_WIDTH, 2)>>>(x,y,z,size);

  printf("%s\n", hipGetErrorString(hipGetLastError()));

  // synchronize 
  hipDeviceSynchronize();

  for (int i = 0; i < size; i++) {
  	for (int j = 0; j < size; j++) {
  		printf("%f ", z[i * size + j]);
  		if (z[i * size + j] != size) {
  			printf("Error at z[%d][%d]: %f\n", i, j, z[i * size + j]);
  		}
    }
    printf("\n");
  }

  	// Freeing memory
	hipFree(x);
	hipFree(y);
	hipFree(z);
  

  return 0;
}
